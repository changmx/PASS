#include "hip/hip_runtime.h"
#include "injection.h"
#include "constant.h"

#include <spdlog/spdlog.h>
#include <spdlog/sinks/stdout_color_sinks.h>
#include <spdlog/sinks/basic_file_sink.h>

#include <iostream>
#include <fstream>
#include <sstream>

#include <random>

Injection::Injection(const Parameter& para, int input_beamId, Bunch& Bunch) {
	std::cout << "pointer 2 " << std::hex << Bunch.dev_bunch << std::endl;
	std::cout << "pointer 3 " << std::hex << dev_bunch << std::endl;

	dev_bunch = Bunch.dev_bunch;
	std::cout << "pointer 4 " << std::hex << dev_bunch << std::endl;

	Np = Bunch.Np;

	alphax = Bunch.alphax;
	alphay = Bunch.alphay;
	betax = Bunch.betax;
	betay = Bunch.betay;
	gammax = Bunch.gammax;
	gammay = Bunch.gammay;

	emitx = Bunch.emitx;
	emity = Bunch.emity;

	sigmaz = Bunch.sigmaz;
	dp = Bunch.dp;

	beamId = input_beamId;
	bunchId = Bunch.bunchId;

	dir_load_distribution = para.dir_load_distribution;
	dir_save_distribution = para.dir_output_distribution;
	hourMinSec = para.hourMinSec;

	beam_name = para.beam_name[input_beamId];

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	std::string key_bunch = "bunch" + std::to_string(Bunch.bunchId);

	try
	{
		s = data.at("Sequence").at("Injection").at("S (m)");
		name = data.at("Sequence").at("Injection").at("Command");

		injection_mode = data.at("Sequence").at("Injection").at(key_bunch).at("Mode");
		dist_transverse = data.at("Sequence").at("Injection").at(key_bunch).at("Transverse dist");
		dist_logitudinal = data.at("Sequence").at("Injection").at(key_bunch).at("Logitudinal dist");

		is_offset_x = data.at("Sequence").at("Injection").at(key_bunch).at("Offset x").at("Is offset");
		is_offset_y = data.at("Sequence").at("Injection").at(key_bunch).at("Offset y").at("Is offset");

		offset_x = data.at("Sequence").at("Injection").at(key_bunch).at("Offset x").at("Offset (m)");
		offset_y = data.at("Sequence").at("Injection").at(key_bunch).at("Offset y").at("Offset (m)");

		is_load_dist = data.at("Sequence").at("Injection").at(key_bunch).at("Is load distribution");
		filename_load_dist = data.at("Sequence").at("Injection").at(key_bunch).at("Name of loaded file");

		is_save_initial_dist = data.at("Sequence").at("Injection").at(key_bunch).at("Is save initial distribution");

		for (size_t i = 0; i < data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns").size(); i++)
		{
			inject_turns.push_back(data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns")[i]);
		}
	}
	catch (json::exception e)
	{
		//std::cout << e.what() << std::endl;
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}
}

void Injection::execute() {
	auto logger = spdlog::get("logger");

	logger->debug("Injection action");

	if (injection_mode == "1turn1time")
	{
		logger->debug("Start: 1-turn and 1-time injection");

		if (is_load_dist)
		{
			load_distribution();
		}
		else if (dist_transverse == "kv")
		{
			generate_KV_distribution();
		}
		else
		{
			logger->error("Sorry, we don't support distribution type {}.", dist_transverse);
			std::exit(EXIT_FAILURE);
		}


		if (is_save_initial_dist)
		{
			save_initial_distribution();
		}

	}
	else if (injection_mode == "1turnxtime")
	{
		logger->error("Sorry, we don't support: 1-turn and multi-time injection.");
		std::exit(EXIT_FAILURE);
	}
	else if (injection_mode == "xturnxtime")
	{
		logger->error("Sorry, we don't support: multi-turn and multi-time injection.");
		std::exit(EXIT_FAILURE);
	}
	else
	{
		logger->error("Input wrong injection mode value: {}.", injection_mode);
	}
}

void Injection::load_distribution() {

	std::filesystem::path dist_path = dir_load_distribution / filename_load_dist;

	if (std::filesystem::exists(dist_path))
	{
		if (filename_load_dist.find(beam_name) == std::string::npos)
			spdlog::get("logger")->warn("Please be careful to confirm that the file is {} distribution: {}.", beam_name, dist_path.string());
		if (filename_load_dist.find(dist_transverse) == std::string::npos)
			spdlog::get("logger")->warn("Please be careful to confirm that the file is {} distribution: {}.", dist_transverse, dist_path.string());
		if (filename_load_dist.find(std::to_string(Np)) == std::string::npos)
			spdlog::get("logger")->warn("Please be careful to confirm that the file contain {} particles: {}.", Np, dist_path.string());

		spdlog::get("logger")->info("... loading distribution file: {}", dist_path.string());

		Particle* host_bunch = new Particle[Np];

		std::ifstream input(dist_path);

		std::string line;
		int j = 0;

		double a[7];
		std::string tmp;
		int row = 0;
		int skiprows = 0;
		while (std::getline(input, line))
		{
			std::stringstream sline(line);
			//std::cout << line << std::endl;
			int k = 0;
			if (row != skiprows)
			{
				while (std::getline(sline, tmp, ','))
				{
					//std::cout << tmp << std::endl;
					a[k] = std::stod(tmp);
					//std::cout << j << a[j] << std::endl;
					++k;
				}
				//std::cout << a[0] << "," << a[1] << std::endl;
				//spdlog::get("logger")->debug("row [{}] a[0] = {}, a[1] = {}", row, a[0], a[1]);

				int offset = j;
				host_bunch[offset].x = a[0];
				host_bunch[offset].px = a[1];
				host_bunch[offset].y = a[2];
				host_bunch[offset].py = a[3];
				host_bunch[offset].z = a[4];
				host_bunch[offset].pz = a[5];
				host_bunch[offset].tag = a[6];

				j++;
			}
			++row;
		}

		if (j != (Np - 1))
		{
			spdlog::get("logger")->warn("We only load {}/{} particles from file {}.", j, Np, dist_path.string());
		}

		input.close();

		//Particle* dev_bunch2;
		//callCuda(hipMalloc(&dev_bunch2, Np * sizeof(double)));
		callCuda(hipMemcpy(dev_bunch, host_bunch, Np * sizeof(Particle), hipMemcpyHostToDevice));

		delete[] host_bunch;
		//callCuda(hipFree(dev_bunch2));

		spdlog::get("logger")->info("... distribution file has been loadded successfully.");
	}
	else
	{
		spdlog::get("logger")->error("We don't find distribution file: {}.", dist_path.string());
		std::exit(EXIT_FAILURE);
	}
}


void Injection::generate_KV_distribution() {

	/* This menthod is derived from "Particle - in - cell code BEAMPATH for beam dynamics simulations in linear accelerators and beamlines"*/
	// The two beams shoule have different seed values to generate different random values.
	// This is 4-D generator.

	spdlog::get("logger")->info("... generating initial KV distribution of beam-{} bunch-{}.", beam_name, bunchId);

	double emittence_x = emitx;
	double emittence_y = emity;
	double alpha_x_twiss = alphax;
	double alpha_y_twiss = alphay;
	double beta_x_twiss = betax;
	double beta_y_twiss = betay;
	double gamma_x_twiss = gammax;
	double gamma_y_twiss = gammay;

	double sigmax = sqrt(emittence_x * betax);
	double sigmay = sqrt(emittence_y * betay);

	// [-1��, 1��] = 0.6826894921370859, [-4��, 4��] = 0.9999366575163338
	// [-2��, 2��] = 0.9544997361036416, [-5��, 5��] = 0.9999994266968562
	// [-3��, 3��] = 0.9973002039367398, [-6��, 6��] = 0.9999999980268246
	double x_max = 4 * sigmax;
	double x_min = -4 * sigmax;
	double y_max = 4 * sigmay;
	double y_min = -4 * sigmay;

	int rank = 0;
	int i = bunchId;
	int beam_label = beamId;

	std::default_random_engine e1;
	e1.seed(time(NULL) + rank * 10000 + (i + 1) * 100 + beam_label);
	std::uniform_real_distribution<> u1(1e-15, 1.0 - 1e-15);

	std::default_random_engine e2;
	e2.seed(time(NULL) + rank * 10000 + (i + 1) * 100 + 10 + beam_label);
	std::uniform_real_distribution<> u2(0, 1);

	Particle* host_bunch = new Particle[Np];

	for (int j = 0; j < Np; ++j)
	{
		double nu, x, px, y, py;
		double X1, X2, Y1, Y2;
		double sigma11_x, sigma11_y, sigma12_x, sigma12_y, sigma22_x, sigma22_y;
		double ax, axpx, ay, aypy;
		double zeta_x, zeta_y, zeta_x_square, zeta_y_square;
		double phi_x, phi_y;
		double beta_x, beta_y;
		double pi = PassConstant::PI;

		double random_zeta = u1(e1);
		double random_beta_x = u2(e2);
		double random_beta_y = u2(e2);

		double F = emittence_x;

		nu = emittence_x / emittence_y;

		sigma11_x = emittence_x * beta_x_twiss;
		sigma12_x = -emittence_x * alpha_x_twiss;
		sigma22_x = emittence_x * gamma_x_twiss;

		sigma11_y = emittence_y * beta_y_twiss;
		sigma12_y = -emittence_y * alpha_y_twiss;
		sigma22_y = emittence_y * gamma_y_twiss;

		phi_x = 0.5 * atan2(2 * sigma12_x, sigma22_x - sigma11_x);
		phi_y = 0.5 * atan2(2 * sigma12_y, sigma22_y - sigma11_y);

		X1 = sqrt(2) * emittence_x / sqrt((sigma11_x + sigma22_x) + sqrt(pow((sigma22_x - sigma11_x), 2) + 4 * pow(sigma12_x, 2)));
		X2 = sqrt(2) * emittence_x / sqrt((sigma11_x + sigma22_x) - sqrt(pow((sigma22_x - sigma11_x), 2) + 4 * pow(sigma12_x, 2)));
		Y1 = sqrt(2) * emittence_y / sqrt((sigma11_y + sigma22_y) + sqrt(pow((sigma22_y - sigma11_y), 2) + 4 * pow(sigma12_y, 2)));
		Y2 = sqrt(2) * emittence_y / sqrt((sigma11_y + sigma22_y) - sqrt(pow((sigma22_y - sigma11_y), 2) + 4 * pow(sigma12_y, 2)));

		ax = sqrt((X1 / X2) * pow(cos(phi_x), 2) + (X2 / X1) * pow(sin(phi_x), 2));
		axpx = (X1 / X2 - X2 / X1) * sin(2 * phi_x) / (2 * ax);
		ay = sqrt((Y1 / Y2) * pow(cos(phi_y), 2) + (Y2 / Y1) * pow(sin(phi_y), 2));
		aypy = (Y1 / Y2 - Y2 / Y1) * sin(2 * phi_y) / (2 * ay);

		zeta_x_square = F * random_zeta;
		zeta_x = sqrt(zeta_x_square);
		zeta_y_square = (F - zeta_x_square) / nu;
		zeta_y = sqrt(zeta_y_square);
		beta_x = 2 * pi * random_beta_x;
		beta_y = 2 * pi * random_beta_y;

		x = zeta_x * ax * cos(beta_x) * 2;
		px = zeta_x * (axpx * cos(beta_x) - sin(beta_x) / ax) * 2;
		y = zeta_y * ay * cos(beta_y) * 2;
		py = zeta_y * (aypy * cos(beta_y) - sin(beta_y) / ay) * 2;
		/*x = zeta_x * ax * cos(beta_x);
		px = zeta_x * (axpx * cos(beta_x) - sin(beta_x) / ax);
		y = zeta_y * ay * cos(beta_y);
		py = zeta_y * (aypy * cos(beta_y) - sin(beta_y) / ay);*/

		if (x > x_min && x < x_max && y > y_min && y < y_max)
		{
			host_bunch[j].x = x;
			host_bunch[j].px = px;
			host_bunch[j].y = y;
			host_bunch[j].py = py;
			host_bunch[j].tag = j + 1;
		}
		else
		{
			--j;
		}
	}

	callCuda(hipMemcpy(dev_bunch, host_bunch, Np * sizeof(Particle), hipMemcpyHostToDevice));

	delete[] host_bunch;
	//std::cout << "initial KV distribution of " << beam.beamName << " has been genetated successfully." << std::endl;
	spdlog::get("logger")->info("... initial KV distribution of {} beam bunch-{} has been genetated successfully.", beam_name, bunchId);
}

void Injection::save_initial_distribution() {

	Particle* host_bunch = new Particle[Np];

	callCuda(hipMemcpy(host_bunch, dev_bunch, Np * sizeof(Particle), hipMemcpyDeviceToHost));

	std::filesystem::path path_tmp = dir_save_distribution / (hourMinSec + "_" + dist_transverse + "_" + beam_name +
		"_bunch" + std::to_string(bunchId) + "_" + std::to_string(Np) + "_initial.csv");
	std::ofstream file(path_tmp);

	file << "x" << "," << "px" << "," << "y" << "," << "py" << "," << "z" << "," << "pz" << "," << "tag" << "," << "lostTurn" << std::endl;

	for (int j = 0; j < Np; j++) {
		file << std::setprecision(10)
			<< (host_bunch + j)->x << ","
			<< (host_bunch + j)->px << ","
			<< (host_bunch + j)->y << ","
			<< (host_bunch + j)->py << ","
			<< (host_bunch + j)->z << ","
			<< (host_bunch + j)->pz << ","
			<< (host_bunch + j)->tag << ","
			<< (host_bunch + j)->lostTurn << "\n";
	}
	file.close();
	delete[]host_bunch;

	spdlog::get("logger")->info("... initial {} distribution of {} beam bunch-{} has been saved to {}.", dist_transverse, beam_name, bunchId, path_tmp.string());
}