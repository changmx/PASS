#include "hip/hip_runtime.h"
#include "injection.h"
#include "constant.h"

#include <spdlog/spdlog.h>
#include <spdlog/sinks/stdout_color_sinks.h>
#include <spdlog/sinks/basic_file_sink.h>

#include <iostream>
#include <fstream>
#include <sstream>

#include <random>

Injection::Injection(const Parameter& para, int input_beamId, Bunch& Bunch, std::string obj_name) {
	//std::cout << "pointer 2 " << std::hex << Bunch.dev_bunch << std::endl;
	//std::cout << "pointer 3 " << std::hex << dev_bunch << std::endl;
	name = obj_name;
	dev_particle = Bunch.dev_particle;
	//std::cout << "pointer 4 " << std::hex << dev_bunch << std::endl;

	Np = Bunch.Np;

	beamId = input_beamId;
	bunchId = Bunch.bunchId;

	dir_load_distribution = para.dir_load_distribution;
	dir_save_distribution = para.dir_output_distribution;
	hourMinSec = para.hourMinSec;

	beam_name = para.beam_name[input_beamId];

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	std::string key_bunch = "bunch" + std::to_string(Bunch.bunchId);

	try
	{
		s = data.at("Sequence").at("Injection").at("S (m)");
		//name = data.at("Sequence").at("Injection").at("Command");
		if (fabs(s) > 1e-10)
		{
			spdlog::get("logger")->error("[Injection] The position of injection point (simulation start point) should be 0, but now is : {}.", s);
			std::exit(EXIT_FAILURE);
		}

		if (1 == data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns").size())
		{
			startTurn = data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns")[0];
			endTurn = data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns")[0];
		}
		else if (2 == data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns").size())
		{
			startTurn = data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns")[0];
			endTurn = data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns")[1];
		}
		else
		{
			spdlog::get("logger")->error("[Injection] The number of parameters of 'Inject turns' should be 1 or 2, but now is: {}.",
				data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns").size());
			std::exit(EXIT_FAILURE);
		}

		alphax = data.at("Sequence").at("Injection").at(key_bunch).at("Alpha x");
		alphay = data.at("Sequence").at("Injection").at(key_bunch).at("Alpha y");

		betax = data.at("Sequence").at("Injection").at(key_bunch).at("Beta x (m)");
		betay = data.at("Sequence").at("Injection").at(key_bunch).at("Beta y (m)");

		gammax = (1 + alphax * alphax) / betax;
		gammay = (1 + alphay * alphay) / betay;

		emitx = data.at("Sequence").at("Injection").at(key_bunch).at("Emittance x (m'rad)");
		emity = data.at("Sequence").at("Injection").at(key_bunch).at("Emittance y (m'rad)");

		Dx = data.at("Sequence").at("Injection").at(key_bunch).at("Dx (m)");
		Dpx = data.at("Sequence").at("Injection").at(key_bunch).at("Dpx");

		emitx_norm = emitx * Bunch.gamma * Bunch.beta;
		emity_norm = emity * Bunch.gamma * Bunch.beta;

		sigmaz = data.at("Sequence").at("Injection").at(key_bunch).at("Sigma z (m)");
		dp = data.at("Sequence").at("Injection").at(key_bunch).at("DeltaP/P");

		sigmax = sqrt(betax * emitx);
		sigmay = sqrt(betay * emity);

		sigmapx = sqrt(gammax * emitx);
		sigmapy = sqrt(gammay * emity);

		injection_mode = data.at("Sequence").at("Injection").at(key_bunch).at("Mode");
		dist_transverse = data.at("Sequence").at("Injection").at(key_bunch).at("Transverse dist");
		dist_longitudinal = data.at("Sequence").at("Injection").at(key_bunch).at("Longitudinal dist");

		is_offset_x = data.at("Sequence").at("Injection").at(key_bunch).at("Offset x").at("Is offset");
		is_offset_y = data.at("Sequence").at("Injection").at(key_bunch).at("Offset y").at("Is offset");

		offset_x = data.at("Sequence").at("Injection").at(key_bunch).at("Offset x").at("Offset (m)");
		offset_y = data.at("Sequence").at("Injection").at(key_bunch).at("Offset y").at("Offset (m)");

		is_load_dist = data.at("Sequence").at("Injection").at(key_bunch).at("Is load distribution");
		filename_load_dist = data.at("Sequence").at("Injection").at(key_bunch).at("Name of loaded file");

		is_save_initial_dist = data.at("Sequence").at("Injection").at(key_bunch).at("Is save initial distribution");

		for (size_t i = 0; i < data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns").size(); i++)
		{
			inject_turns.push_back(data.at("Sequence").at("Injection").at(key_bunch).at("Inject turns")[i]);
		}

		if (data.at("Sequence").at("Injection").at(key_bunch).at("Particle coordinate").size() > 0) {
			is_set_specified_coordinate = true;

			for (size_t i = 0; i < data.at("Sequence").at("Injection").at(key_bunch).at("Particle coordinate").size(); i++)
			{
				double x_tmp = data.at("Sequence").at("Injection").at(key_bunch).at("Particle coordinate")[i][0];
				double px_tmp = data.at("Sequence").at("Injection").at(key_bunch).at("Particle coordinate")[i][1];
				double y_tmp = data.at("Sequence").at("Injection").at(key_bunch).at("Particle coordinate")[i][2];
				double py_tmp = data.at("Sequence").at("Injection").at(key_bunch).at("Particle coordinate")[i][3];
				double z_tmp = data.at("Sequence").at("Injection").at(key_bunch).at("Particle coordinate")[i][4];
				double dp_tmp = data.at("Sequence").at("Injection").at(key_bunch).at("Particle coordinate")[i][5];

				specified_coordinate.push_back({ x_tmp, px_tmp, y_tmp, py_tmp, z_tmp, dp_tmp });
			}
		}
	}
	catch (json::exception e)
	{
		//std::cout << e.what() << std::endl;
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}

	Bunch.sigmaz = sigmaz;
	Bunch.dp = dp;

	Bunch.dist_transverse = dist_transverse;
	Bunch.dist_longitudinal = dist_longitudinal;
}

void Injection::execute(int turn) {
	auto logger = spdlog::get("logger");

	//logger->debug("Injection action");

	if ("1turn1time" == injection_mode)
	{
		//logger->debug("Start: 1-turn and 1-time injection");
		if (startTurn != endTurn)
		{
			logger->warn("[Injection] In the 1-turn 1-time injection mode, we only inject 1 turn, but input parameters is from turn {} to turn {}. We will only inject at turn {}.",
				startTurn, endTurn, startTurn);
		}

		if (turn == startTurn)
		{
			if (is_load_dist)
			{
				load_distribution();
			}
			else
			{
				if ("kv" == dist_transverse)
				{
					generate_transverse_KV_distribution();
				}
				else if ("gaussian" == dist_transverse)
				{
					generate_transverse_Gaussian_distribution();
				}
				else if ("uniform" == dist_transverse)
				{
					generate_transverse_uniform_distribution();
				}
				else
				{
					logger->error("[Injection] Sorry, we don't support transverse distribution type {}.", dist_transverse);
					std::exit(EXIT_FAILURE);
				}

				if ("gaussian" == dist_longitudinal)
				{
					generate_longitudinal_Gaussian_distribution();
				}
				else if ("uniform" == dist_longitudinal)
				{
					generate_longitudinal_uniform_distribution();
				}
				else
				{
					logger->error("[Injection] Sorry, we don't support longitudinal distribution type {}.", dist_longitudinal);
					std::exit(EXIT_FAILURE);
				}

				add_Dx();
				add_offset();

			}

			if (is_save_initial_dist)
			{
				save_initial_distribution();
			}
		}
	}

	else if ("1turnxtime" == injection_mode)
	{
		logger->error("[Injection] Sorry, we don't support: 1-turn and multi-time injection.");
		std::exit(EXIT_FAILURE);
	}

	else if ("xturnxtime" == injection_mode)
	{
		logger->error("[Injection] Sorry, we don't support: multi-turn and multi-time injection.");
		std::exit(EXIT_FAILURE);
	}

	else
	{
		logger->error("[Injection] Input wrong injection mode value: {}.", injection_mode);
		std::exit(EXIT_FAILURE);
	}
}

void Injection::load_distribution() {

	std::filesystem::path dist_path = dir_load_distribution / filename_load_dist;

	if (std::filesystem::exists(dist_path))
	{
		if (filename_load_dist.find(beam_name) == std::string::npos)
			spdlog::get("logger")->warn("[Injection] Please be careful to confirm that the file is {} distribution: {}.", beam_name, dist_path.string());
		if (filename_load_dist.find(dist_transverse) == std::string::npos)
			spdlog::get("logger")->warn("[Injection] Please be careful to confirm that the file is {} distribution: {}.", dist_transverse, dist_path.string());
		if (filename_load_dist.find(std::to_string(Np)) == std::string::npos)
			spdlog::get("logger")->warn("[Injection] Please be careful to confirm that the file contain {} particles: {}.", Np, dist_path.string());

		spdlog::get("logger")->info("[Injection] Loading distribution file: {}", dist_path.string());

		Particle host_particle;
		host_particle.mem_allocate_cpu(Np);

		std::ifstream input(dist_path);

		std::string line;
		int j = 0;

		double a[6] = { 0,0,0,0,0,0 };
		int a_tag = 0, a_lostTurn = -1;
		int a_sliceId = 0;
		double a_lostPos = -1.0;
		std::string tmp;
		int row = 0;
		int skiprows = 0;
		while (std::getline(input, line))
		{
			std::stringstream sline(line);
			//std::cout << line << std::endl;
			int k = 0;
			if (row != skiprows)
			{
				while (std::getline(sline, tmp, ','))
				{
					if (k < 6)
					{
						//std::cout << tmp << std::endl;
						a[k] = std::stod(tmp);
						//std::cout << j << a[j] << std::endl;
					}
					else if (k == 6)
					{
						a_tag = std::stoi(tmp);
					}
					else if (k == 7)
					{
						a_sliceId = std::stoi(tmp);
					}
					else if (k == 8)
					{
						a_lostTurn = std::stoi(tmp);
					}
					else if (k == 9)
					{
						a_lostPos = std::stod(tmp);
					}
					++k;
				}
				//std::cout << a[0] << "," << a[1] << std::endl;
				//spdlog::get("logger")->debug("row [{}] a[0] = {}, a[1] = {}", row, a[0], a[1]);

				int offset = j;
				if (offset < Np)
				{
					host_particle.x[offset] = a[0];
					host_particle.px[offset] = a[1];
					host_particle.y[offset] = a[2];
					host_particle.py[offset] = a[3];
					host_particle.z[offset] = a[4];
					host_particle.pz[offset] = a[5];
					host_particle.tag[offset] = a_tag;
					host_particle.sliceId[offset] = a_sliceId;
					host_particle.lostTurn[offset] = a_lostTurn;
					host_particle.lostPos[offset] = a_lostPos;

					j++;
				}

			}
			++row;
		}

		if (j != Np)
		{
			spdlog::get("logger")->warn("[Injection] We only load {}/{} particles from file {}.", j, Np, dist_path.string());
		}

		input.close();

		if (is_set_specified_coordinate) {

			for (size_t i = 0; i < specified_coordinate.size(); i++)
			{
				host_particle.x[i] = specified_coordinate[i][0];
				host_particle.px[i] = specified_coordinate[i][1];
				host_particle.y[i] = specified_coordinate[i][2];
				host_particle.py[i] = specified_coordinate[i][3];
				host_particle.z[i] = specified_coordinate[i][4];
				host_particle.pz[i] = specified_coordinate[i][5];
			}
		}

		particle_copy(dev_particle, host_particle, Np, hipMemcpyHostToDevice, "dist");

		host_particle.mem_free_cpu();

		spdlog::get("logger")->info("[Injection] Distribution file {} has been loadded successfully to {} beam-{} bunch-{}.",
			dist_path.string(), beam_name, beamId, bunchId);
	}
	else
	{
		spdlog::get("logger")->error("[Injection] We don't find distribution file: {}.", dist_path.string());
		std::exit(EXIT_FAILURE);
	}
}


void Injection::generate_transverse_KV_distribution() {

	/* This menthod is derived from "Particle - in - cell code BEAMPATH for beam dynamics simulations in linear accelerators and beamlines"*/
	// The two beams shoule have different seed values to generate different random values.
	// This is 4-D generator.

	spdlog::get("logger")->info("[Injection] The initial transverse KV distribution of {} beam-{} bunch-{} is begin generated ...",
		beam_name, beamId, bunchId);

	double emittence_x = emitx;
	double emittence_y = emity;
	double alpha_x_twiss = alphax;
	double alpha_y_twiss = alphay;
	double beta_x_twiss = betax;
	double beta_y_twiss = betay;
	double gamma_x_twiss = gammax;
	double gamma_y_twiss = gammay;

	double sigmax = sqrt(emittence_x * betax);
	double sigmay = sqrt(emittence_y * betay);

	// [-1��, 1��] = 0.6826894921370859, [-4��, 4��] = 0.9999366575163338
	// [-2��, 2��] = 0.9544997361036416, [-5��, 5��] = 0.9999994266968562
	// [-3��, 3��] = 0.9973002039367398, [-6��, 6��] = 0.9999999980268246
	double x_max = 4 * sigmax;
	double x_min = -4 * sigmax;
	double y_max = 4 * sigmay;
	double y_min = -4 * sigmay;

	int i = bunchId;
	int beam_label = beamId;

	std::default_random_engine e1;
	e1.seed(curTime + beam_label * 10000019 + (callTime++) * 1000 + (i + 1) * 1);
	std::uniform_real_distribution<> u1(1e-15, 1.0 - 1e-15);

	std::default_random_engine e2;
	e2.seed(curTime + beam_label * 10000019 + (callTime++) * 1000 + (i + 1) * 1);
	std::uniform_real_distribution<> u2(0, 1);

	Particle host_particle;
	host_particle.mem_allocate_cpu(Np);

	for (int j = 0; j < Np; ++j)
	{
		double nu, x, px, y, py;
		double X1, X2, Y1, Y2;
		double sigma11_x, sigma11_y, sigma12_x, sigma12_y, sigma22_x, sigma22_y;
		double ax, axpx, ay, aypy;
		double zeta_x, zeta_y, zeta_x_square, zeta_y_square;
		double phi_x, phi_y;
		double beta_x, beta_y;
		double pi = PassConstant::PI;

		double random_zeta = u1(e1);
		double random_beta_x = u2(e2);
		double random_beta_y = u2(e2);

		double F = emittence_x;

		nu = emittence_x / emittence_y;

		sigma11_x = emittence_x * beta_x_twiss;
		sigma12_x = -emittence_x * alpha_x_twiss;
		sigma22_x = emittence_x * gamma_x_twiss;

		sigma11_y = emittence_y * beta_y_twiss;
		sigma12_y = -emittence_y * alpha_y_twiss;
		sigma22_y = emittence_y * gamma_y_twiss;

		phi_x = 0.5 * atan2(2 * alpha_x_twiss, gamma_x_twiss - beta_x_twiss);	// https://agenda.linearcollider.org/event/6258/contributions/29168/attachments/24202/37474/linear_dynamics.pdf
		phi_y = 0.5 * atan2(2 * alpha_y_twiss, gamma_y_twiss - beta_y_twiss);

		X1 = sqrt(2) * emittence_x / sqrt((sigma11_x + sigma22_x) + sqrt(pow((sigma22_x - sigma11_x), 2) + 4 * pow(sigma12_x, 2)));
		X2 = sqrt(2) * emittence_x / sqrt((sigma11_x + sigma22_x) - sqrt(pow((sigma22_x - sigma11_x), 2) + 4 * pow(sigma12_x, 2)));
		Y1 = sqrt(2) * emittence_y / sqrt((sigma11_y + sigma22_y) + sqrt(pow((sigma22_y - sigma11_y), 2) + 4 * pow(sigma12_y, 2)));
		Y2 = sqrt(2) * emittence_y / sqrt((sigma11_y + sigma22_y) - sqrt(pow((sigma22_y - sigma11_y), 2) + 4 * pow(sigma12_y, 2)));

		ax = sqrt((X1 / X2) * pow(cos(phi_x), 2) + (X2 / X1) * pow(sin(phi_x), 2));
		axpx = (X1 / X2 - X2 / X1) * sin(2 * phi_x) / (2 * ax);
		ay = sqrt((Y1 / Y2) * pow(cos(phi_y), 2) + (Y2 / Y1) * pow(sin(phi_y), 2));
		aypy = (Y1 / Y2 - Y2 / Y1) * sin(2 * phi_y) / (2 * ay);

		zeta_x_square = F * random_zeta;
		zeta_x = sqrt(zeta_x_square);
		zeta_y_square = (F - zeta_x_square) / nu;
		zeta_y = sqrt(zeta_y_square);
		beta_x = 2 * pi * random_beta_x;
		beta_y = 2 * pi * random_beta_y;

		x = zeta_x * ax * cos(beta_x) * 2;
		px = zeta_x * (axpx * cos(beta_x) - sin(beta_x) / ax) * 2;
		y = zeta_y * ay * cos(beta_y) * 2;
		py = zeta_y * (aypy * cos(beta_y) - sin(beta_y) / ay) * 2;
		/*x = zeta_x * ax * cos(beta_x);
		px = zeta_x * (axpx * cos(beta_x) - sin(beta_x) / ax);
		y = zeta_y * ay * cos(beta_y);
		py = zeta_y * (aypy * cos(beta_y) - sin(beta_y) / ay);*/

		if (x > x_min && x < x_max && y > y_min && y < y_max)
		{
			host_particle.x[j] = x;
			host_particle.px[j] = px;
			host_particle.y[j] = y;
			host_particle.py[j] = py;
			host_particle.tag[j] = j + 1;
			host_particle.sliceId[j] = 0;
			host_particle.lostTurn[j] = -1;
			host_particle.lostPos[j] = -1.0;
		}
		else
		{
			--j;
		}
	}

	if (is_set_specified_coordinate) {

		for (size_t i = 0; i < specified_coordinate.size(); i++)
		{
			host_particle.x[i] = specified_coordinate[i][0];
			host_particle.px[i] = specified_coordinate[i][1];
			host_particle.y[i] = specified_coordinate[i][2];
			host_particle.py[i] = specified_coordinate[i][3];
		}
	}

	//callCuda(hipMemcpy(dev_bunch, host_bunch, Np * sizeof(Particle), hipMemcpyHostToDevice));
	particle_copy(dev_particle, host_particle, Np, hipMemcpyHostToDevice, "dist");

	host_particle.mem_free_cpu();
	//std::cout << "initial KV distribution of " << beam.beamName << " has been genetated successfully." << std::endl;
	spdlog::get("logger")->info("[Injection] The initial transverse KV distribution of {} beam-{} bunch-{} has been genetated successfully.",
		beam_name, beamId, bunchId);
}


void Injection::generate_transverse_Gaussian_distribution() {

	spdlog::get("logger")->info("[Injection] The initial transverse Gaussian distribution of {} beam-{} bunch-{} is begin generated ...",
		beam_name, beamId, bunchId);

	double emittence_x = emitx;
	double emittence_y = emity;
	double alpha_x_twiss = alphax;
	double alpha_y_twiss = alphay;
	double beta_x_twiss = betax;
	double beta_y_twiss = betay;
	double gamma_x_twiss = gammax;
	double gamma_y_twiss = gammay;

	double sigmax = sqrt(emittence_x * betax);
	double sigmay = sqrt(emittence_y * betay);

	// [-1��, 1��] = 0.6826894921370859, [-4��, 4��] = 0.9999366575163338
	// [-2��, 2��] = 0.9544997361036416, [-5��, 5��] = 0.9999994266968562
	// [-3��, 3��] = 0.9973002039367398, [-6��, 6��] = 0.9999999980268246
	double x_max = 4 * sigmax;
	double x_min = -4 * sigmax;
	double y_max = 4 * sigmay;
	double y_min = -4 * sigmay;

	int i = bunchId;
	int beam_label = beamId;

	std::default_random_engine e1;
	e1.seed(curTime + beam_label * 10000019 + (callTime++) * 1000 + (i + 1) * 1);
	std::uniform_real_distribution<> u1(1e-15, 1.0 - 1e-15);

	Particle host_particle;
	host_particle.mem_allocate_cpu(Np);
	//Particle* host_bunch;
	//hipHostAlloc((void**)&host_bunch, Np * sizeof(Particle), hipHostMallocDefault);


	for (int j = 0; j < Np; ++j)
	{

		double x, px, y, py;
		double Xm, thetaXm, Ym, thetaYm;
		double a_x, a_y, u_x, u_y, v_x, v_y;
		double alpha_x, alpha_y, chi_x, chi_y;
		double pi = PassConstant::PI;

		double random_s1_x = u1(e1); //range (0,1)
		double random_s1_y = u1(e1); //range (0,1)
		double random_s2_x = u1(e1); //range (0,1)
		double random_s2_y = u1(e1); //range (0,1)

		Xm = 2 * sqrt(emittence_x * beta_x_twiss);
		thetaXm = 2 * sqrt(emittence_x * gamma_x_twiss);
		a_x = sqrt(2) / 2 * sqrt(-log(random_s1_x));
		alpha_x = 2 * pi * random_s2_x;
		chi_x = -1 * atan(alpha_x_twiss);
		u_x = a_x * cos(alpha_x);
		v_x = a_x * sin(alpha_x);
		x = Xm * u_x;
		px = thetaXm * (u_x * sin(chi_x) + v_x * cos(chi_x));

		Ym = 2 * sqrt(emittence_y * beta_y_twiss);
		thetaYm = 2 * sqrt(emittence_y * gamma_y_twiss);
		a_y = sqrt(2) / 2 * sqrt(-log(random_s1_y));
		alpha_y = 2 * pi * random_s2_y;
		chi_y = -1 * atan(alpha_y_twiss);
		u_y = a_y * cos(alpha_y);
		v_y = a_y * sin(alpha_y);
		y = Ym * u_y;
		py = thetaYm * (u_y * sin(chi_y) + v_y * cos(chi_y));

		if (x > x_min && x < x_max && y > y_min && y < y_max)
		{
			host_particle.x[j] = x;
			host_particle.px[j] = px;
			host_particle.y[j] = y;
			host_particle.py[j] = py;
			host_particle.tag[j] = j + 1;
			host_particle.sliceId[j] = 0;
			host_particle.lostTurn[j] = -1;
			host_particle.lostPos[j] = -1.0;
		}
		else
		{
			--j;
		}
		/*std::cout << "tag: " << beam.tag(i) << std::endl;
		std::cout << beam.x(i) << " " << beam.px(i) << " " << beam.y(i) << " " << beam.py(i) << std::endl;*/
	}

	if (is_set_specified_coordinate) {

		for (size_t i = 0; i < specified_coordinate.size(); i++)
		{
			host_particle.x[i] = specified_coordinate[i][0];
			host_particle.px[i] = specified_coordinate[i][1];
			host_particle.y[i] = specified_coordinate[i][2];
			host_particle.py[i] = specified_coordinate[i][3];
		}
	}

	//callCuda(hipMemcpy(dev_bunch, host_bunch, Np * sizeof(Particle), hipMemcpyHostToDevice));
	particle_copy(dev_particle, host_particle, Np, hipMemcpyHostToDevice, "dist");

	host_particle.mem_free_cpu();
	//hipHostFree(host_bunch);
	//std::cout << "initial Gaussian distribution of " << beam.beamName << " has been genetated successfully." << std::endl;
	spdlog::get("logger")->info("[Injection] The initial transverse Gaussian distribution of {} beam-{} bunch-{} has been genetated successfully.",
		beam_name, beamId, bunchId);

}

void Injection::generate_transverse_uniform_distribution() {

	spdlog::get("logger")->info("[Injection] The initial transverse uniform distribution of {} beam-{} bunch-{} is begin generated ...",
		beam_name, beamId, bunchId);

	double emittence_x = emitx;
	double emittence_y = emity;
	double alpha_x_twiss = alphax;
	double alpha_y_twiss = alphay;
	double beta_x_twiss = betax;
	double beta_y_twiss = betay;
	double gamma_x_twiss = gammax;
	double gamma_y_twiss = gammay;

	double sigmax = sqrt(emittence_x * betax);
	double sigmay = sqrt(emittence_y * betay);

	// [-1��, 1��] = 0.6826894921370859, [-4��, 4��] = 0.9999366575163338
	// [-2��, 2��] = 0.9544997361036416, [-5��, 5��] = 0.9999994266968562
	// [-3��, 3��] = 0.9973002039367398, [-6��, 6��] = 0.9999999980268246
	double x_max = 4 * sigmax;
	double x_min = -4 * sigmax;
	double y_max = 4 * sigmay;
	double y_min = -4 * sigmay;

	int i = bunchId;
	int beam_label = beamId;

	std::default_random_engine e1;
	e1.seed(curTime + beam_label * 10000019 + (callTime++) * 1000 + (i + 1) * 1);
	std::uniform_real_distribution<> u1(1e-15, 1.0 - 1e-15);

	Particle host_particle;
	host_particle.mem_allocate_cpu(Np);

	for (int j = 0; j < Np; ++j)
	{

		double m = 1;
		double x, px, y, py;
		double Xm, thetaXm, Ym, thetaYm;
		double Xl, Xlpx, Yl, Ylpy;
		double a_x, a_y, u_x, u_y, v_x, v_y;
		double alpha_x, alpha_y, chi_x, chi_y;
		double pi = PassConstant::PI;

		double random_s1_x = u1(e1); //range (0,1)
		double random_s2_x = u1(e1); //range (0,1)
		double random_s1_y = u1(e1); //range (0,1)
		double random_s2_y = u1(e1); //range (0,1)

		Xm = 2 * sqrt(emittence_x * beta_x_twiss);
		thetaXm = 2 * sqrt(emittence_x * gamma_x_twiss);
		Xl = sqrt((m + 1) / 2) * Xm;
		Xlpx = sqrt((m + 1) / 2) * thetaXm;
		a_x = sqrt(1 - pow(random_s1_x, 1 / m));
		alpha_x = 2 * pi * random_s2_x;
		chi_x = -1 * atan(alpha_x_twiss);
		u_x = a_x * cos(alpha_x);
		v_x = a_x * sin(alpha_x);

		x = Xl * u_x;
		px = Xlpx * (u_x * sin(chi_x) + v_x * cos(chi_x));
		//printf("Xm = %f, thetaXm = %f, Xl = %f, Xlpx = %f, alphaX = %f, ux = %f, vx = %f, x = %f, px = %f\n", Xm, thetaXm, Xl, Xlpx, alpha_x, u_x, v_x, x, px);

		Ym = 2 * sqrt(emittence_y * beta_y_twiss);
		thetaYm = 2 * sqrt(emittence_y * gamma_y_twiss);
		Yl = sqrt((m + 1) / 2) * Ym;
		Ylpy = sqrt((m + 1) / 2) * thetaYm;
		a_y = sqrt(1 - pow(random_s1_y, 1 / m));
		alpha_y = 2 * pi * random_s2_y;
		chi_y = -1 * atan(alpha_y_twiss);
		u_y = a_y * cos(alpha_y);
		v_y = a_y * sin(alpha_y);

		y = Yl * u_y;
		py = Ylpy * (u_y * sin(chi_y) + v_y * cos(chi_y));

		if (x > x_min && x < x_max && y > y_min && y < y_max)
		{
			host_particle.x[j] = x;
			host_particle.px[j] = px;
			host_particle.y[j] = y;
			host_particle.py[j] = py;
			host_particle.tag[j] = j + 1;
			host_particle.sliceId[j] = 0;
			host_particle.lostTurn[j] = -1;
			host_particle.lostPos[j] = -1.0;
		}
		else
		{
			--j;
		}
		/*std::cout << "tag: " << beam.tag(i) << std::endl;
		std::cout << beam.x(i) << " " << beam.px(i) << " " << beam.y(i) << " " << beam.py(i) << std::endl;*/
	}

	if (is_set_specified_coordinate) {

		for (size_t i = 0; i < specified_coordinate.size(); i++)
		{
			host_particle.x[i] = specified_coordinate[i][0];
			host_particle.px[i] = specified_coordinate[i][1];
			host_particle.y[i] = specified_coordinate[i][2];
			host_particle.py[i] = specified_coordinate[i][3];
		}
	}

	//callCuda(hipMemcpy(dev_bunch, host_bunch, Np * sizeof(Particle), hipMemcpyHostToDevice));
	particle_copy(dev_particle, host_particle, Np, hipMemcpyHostToDevice, "dist");

	host_particle.mem_free_cpu();
	//std::cout << "initial Uniform distribution of " << beam.beamName << " has been genetated successfully." << std::endl;
	spdlog::get("logger")->info("[Injection] The initial transverse uniform distribution of {} beam-{} bunch-{} has been genetated successfully.",
		beam_name, beamId, bunchId);
}


void Injection::generate_longitudinal_Gaussian_distribution() {

	//	Generate particle's z position and momentum.
	//	Here we think the correlation coefficient of 2D Gaussian distribution rho = 0

	spdlog::get("logger")->info("[Injection] The initial longitudinal Gaussian distribution of {} beam-{} bunch-{} is begin generated ...",
		beam_name, beamId, bunchId);

	int i = bunchId;
	int beam_label = beamId;

	double sigma_z = sigmaz;
	double sigma_pz = dp;
	//double rho = 0;
	double tmp_z, tmp_pz;

	std::default_random_engine e1;
	e1.seed(curTime + beam_label * 10000019 + (callTime++) * 1000 + (i + 1) * 1);
	std::normal_distribution<> n1(0, sigma_z);

	std::default_random_engine e2;
	e2.seed(curTime + beam_label * 10000019 + (callTime++) * 1000 + (i + 1) * 1);
	std::normal_distribution<> n2(0, sigma_pz);

	Particle host_particle;
	host_particle.mem_allocate_cpu(Np);

	//callCuda(hipMemcpy(host_bunch, dev_bunch, Np * sizeof(Particle), hipMemcpyDeviceToHost));
	particle_copy(host_particle, dev_particle, Np, hipMemcpyDeviceToHost, "dist");

	for (int j = 0; j < Np; ++j)
	{
		tmp_z = n1(e1);
		tmp_pz = n2(e2);

		if (tmp_z >= (-4 * sigma_z) && tmp_z <= (4 * sigma_z))
		{
			host_particle.z[j] = tmp_z;
			host_particle.pz[j] = tmp_pz;
		}
		else
		{
			--j;
		}
	}

	if (is_set_specified_coordinate) {

		for (size_t i = 0; i < specified_coordinate.size(); i++)
		{
			host_particle.z[i] = specified_coordinate[i][4];
			host_particle.pz[i] = specified_coordinate[i][5];
		}
	}

	//callCuda(hipMemcpy(dev_bunch, host_bunch, Np * sizeof(Particle), hipMemcpyHostToDevice));
	particle_copy(dev_particle, host_particle, Np, hipMemcpyHostToDevice, "dist");

	host_particle.mem_free_cpu();
	spdlog::get("logger")->info("[Injection] The initial longitudinal Gaussian distribution of {} beam-{} bunch-{} has been genetated successfully.",
		beam_name, beamId, bunchId);

}


void Injection::generate_longitudinal_uniform_distribution() {

	//	Generate particle's z position and momentum.
	//	Here we think z follows a uniform distribution and pz follows a Gaussian distribution.
	//	The uniform distribution of z ranges from 0 to sigmaz.

	spdlog::get("logger")->info("[Injection] The initial longitudinal uniform distribution of {} beam-{} bunch-{} is begin generated ...",
		beam_name, beamId, bunchId);

	int i = bunchId;
	int beam_label = beamId;

	double sigma_z = sigmaz;	// Acctually, this is the range of uniform distribution, not RMS value
	double sigma_pz = dp;
	//double rho = 0;
	double tmp_z, tmp_pz;

	std::default_random_engine e1;
	e1.seed(curTime + beam_label * 10000019 + (callTime++) * 1000 + (i + 1) * 1);
	std::uniform_real_distribution<> n1(-0.5 * sigma_z, 0.5 * sigma_z);

	std::default_random_engine e2;
	e2.seed(curTime + beam_label * 10000019 + (callTime++) * 1000 + (i + 1) * 1);
	std::normal_distribution<> n2(0, sigma_pz);

	Particle host_particle;
	host_particle.mem_allocate_cpu(Np);

	//callCuda(hipMemcpy(host_bunch, dev_bunch, Np * sizeof(Particle), hipMemcpyDeviceToHost));
	particle_copy(host_particle, dev_particle, Np, hipMemcpyDeviceToHost, "dist");

	for (int j = 0; j < Np; ++j)
	{
		tmp_z = n1(e1);
		tmp_pz = n2(e2);

		if (tmp_z >= (-0.5 * sigma_z) && tmp_z <= (0.5 * sigma_z))
		{
			host_particle.z[j] = tmp_z;
			host_particle.pz[j] = tmp_pz;
		}
		else
		{
			--j;
		}
	}

	if (is_set_specified_coordinate) {

		for (size_t i = 0; i < specified_coordinate.size(); i++)
		{
			host_particle.z[i] = specified_coordinate[i][4];
			host_particle.pz[i] = specified_coordinate[i][5];
		}
	}

	//callCuda(hipMemcpy(dev_bunch, host_bunch, Np * sizeof(Particle), hipMemcpyHostToDevice));
	particle_copy(dev_particle, host_particle, Np, hipMemcpyHostToDevice, "dist");

	host_particle.mem_free_cpu();
	spdlog::get("logger")->info("[Injection] The initial longitudinal uniform distribution of {} beam-{} bunch-{} has been genetated successfully.",
		beam_name, beamId, bunchId);

}


void Injection::save_initial_distribution() {

	Particle host_particle;
	host_particle.mem_allocate_cpu(Np);

	//callCuda(hipMemcpy(host_bunch, dev_bunch, Np * sizeof(Particle), hipMemcpyDeviceToHost));
	particle_copy(host_particle, dev_particle, Np, hipMemcpyDeviceToHost, "dist");

	std::filesystem::path path_tmp = dir_save_distribution / (hourMinSec + "_beam" + std::to_string(beamId) + "_" + beam_name + "_bunch" + std::to_string(bunchId)
		+ "_" + std::to_string(Np) + "_hor_" + dist_transverse + "_longi_" + dist_longitudinal
		+ "_Dx_" + std::to_string(Dx) + "_injection.csv");
	std::ofstream file(path_tmp);

	file << "x" << "," << "px" << "," << "y" << "," << "py" << "," << "z" << "," << "pz" << ","
		<< "tag" << "," << "sliceId" << "," << "lostTurn" << "," << "lostPos" << std::endl;

	for (int j = 0; j < Np; j++) {
		file << std::setprecision(10)
			<< host_particle.x[j] << ","
			<< host_particle.px[j] << ","
			<< host_particle.y[j] << ","
			<< host_particle.py[j] << ","
			<< host_particle.z[j] << ","
			<< host_particle.pz[j] << ","
			<< host_particle.tag[j] << ","
			<< host_particle.sliceId[j] << ","
			<< host_particle.lostTurn[j] << ","
			<< host_particle.lostPos[j] << "\n";
	}
	file.close();
	host_particle.mem_free_cpu();

	spdlog::get("logger")->info("[Injection] Initial {} distribution of {} beam-{} bunch-{} has been saved to {}.",
		dist_transverse, beam_name, beamId, bunchId, path_tmp.string());

}

void Injection::add_Dx() {

	if (fabs(Dx) < 1e-10 && fabs(Dpx) < 1e-10)
	{
		return;
	}

	spdlog::get("logger")->info("[Injection] Dx = {}, Dpx = {} of {} beam-{} bunch-{} is begin added ...",
		Dx, Dpx, beam_name, beamId, bunchId);

	Particle host_particle;
	host_particle.mem_allocate_cpu(Np);

	//callCuda(hipMemcpy(host_bunch, dev_bunch, Np * sizeof(Particle), hipMemcpyDeviceToHost));
	particle_copy(host_particle, dev_particle, Np, hipMemcpyDeviceToHost, "dist");

	for (int j = 0; j < Np; ++j)
	{
		host_particle.x[j] += Dx * host_particle.pz[j];
		host_particle.px[j] += Dpx * host_particle.pz[j];
	}

	//callCuda(hipMemcpy(dev_bunch, host_bunch, Np * sizeof(Particle), hipMemcpyHostToDevice));
	particle_copy(dev_particle, host_particle, Np, hipMemcpyHostToDevice, "dist");

	host_particle.mem_free_cpu();
	spdlog::get("logger")->info("[Injection] Dispersion = {} of {} beam-{} bunch-{} has been genetated successfully.",
		Dx, beam_name, beamId, bunchId);
}


void Injection::add_offset() {

	if (is_offset_x)
	{
		spdlog::get("logger")->info("[Injection] Offset x = {} of {} beam-{} bunch-{} is begin added ...",
			offset_x, beam_name, beamId, bunchId);

		Particle host_particle;
		host_particle.mem_allocate_cpu(Np);

		//callCuda(hipMemcpy(host_bunch, dev_bunch, Np * sizeof(Particle), hipMemcpyDeviceToHost));
		particle_copy(host_particle, dev_particle, Np, hipMemcpyDeviceToHost, "dist");

		for (int j = 0; j < Np; ++j)
		{
			host_particle.x[j] += offset_x;
		}

		//callCuda(hipMemcpy(dev_bunch, host_bunch, Np * sizeof(Particle), hipMemcpyHostToDevice));
		particle_copy(dev_particle, host_particle, Np, hipMemcpyHostToDevice, "dist");

		host_particle.mem_free_cpu();
		spdlog::get("logger")->info("[Injection] Offset x = {} of {} beam-{} bunch-{} has been genetated successfully.",
			offset_x, beam_name, beamId, bunchId);
	}

	if (is_offset_y)
	{
		spdlog::get("logger")->info("[Injection] Offset y = {} of {} beam-{} bunch-{} is begin added ...",
			offset_y, beam_name, beamId, bunchId);

		Particle host_particle;
		host_particle.mem_allocate_cpu(Np);

		//callCuda(hipMemcpy(host_bunch, dev_bunch, Np * sizeof(Particle), hipMemcpyDeviceToHost));
		particle_copy(host_particle, dev_particle, Np, hipMemcpyDeviceToHost, "dist");

		for (int j = 0; j < Np; ++j)
		{
			host_particle.y[j] += offset_y;
		}

		//callCuda(hipMemcpy(dev_bunch, host_bunch, Np * sizeof(Particle), hipMemcpyHostToDevice));
		particle_copy(dev_particle, host_particle, Np, hipMemcpyHostToDevice, "dist");

		host_particle.mem_free_cpu();
		spdlog::get("logger")->info("[Injection] Offset y = {} of {} beam-{} bunch-{} has been genetated successfully.",
			offset_y, beam_name, beamId, bunchId);
	}
}

void Injection::print_config() {


}