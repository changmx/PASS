#include "hip/hip_runtime.h"
#include "twiss.h"
#include "parameter.h"
#include "constant.h"

#include <fstream>
#include <hip/hip_runtime.h>

Twiss::Twiss(const Parameter& para, int input_beamId, const Bunch& Bunch, std::string obj_name, const ParallelPlan1d& plan1d, TimeEvent& timeevent)
	:simTime(timeevent), bunchRef(Bunch) {

	name = obj_name;
	dev_particle = Bunch.dev_particle;

	thread_x = plan1d.get_threads_per_block();
	block_x = plan1d.get_blocks_x();

	Np = Bunch.Np;
	circumference = para.circumference;

	gamma = Bunch.gamma;
	gammat = Bunch.gammat;
	sigmaz = Bunch.sigmaz;
	dp = Bunch.dp;

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	//std::string key_bunch = "bunch" + std::to_string(Bunch.bunchId);

	try
	{
		s = data.at("Sequence").at(obj_name).at("S (m)");
		s_previous = data.at("Sequence").at(obj_name).at("S previous (m)");

		alphax = data.at("Sequence").at(obj_name).at("Alpha x");
		alphay = data.at("Sequence").at(obj_name).at("Alpha y");
		alphax_previous = data.at("Sequence").at(obj_name).at("Alpha x previous");
		alphay_previous = data.at("Sequence").at(obj_name).at("Alpha y previous");

		betax = data.at("Sequence").at(obj_name).at("Beta x (m)");
		betay = data.at("Sequence").at(obj_name).at("Beta y (m)");
		betax_previous = data.at("Sequence").at(obj_name).at("Beta x previous (m)");
		betay_previous = data.at("Sequence").at(obj_name).at("Beta y previous (m)");

		mux = data.at("Sequence").at(obj_name).at("Mu x");
		muy = data.at("Sequence").at(obj_name).at("Mu y");
		mux_previous = data.at("Sequence").at(obj_name).at("Mu x previous");
		muy_previous = data.at("Sequence").at(obj_name).at("Mu y previous");

		Dx = data.at("Sequence").at(obj_name).at("Dx (m)");
		Dx_previous = data.at("Sequence").at(obj_name).at("Dx (m) previous");

		Dpx = data.at("Sequence").at(obj_name).at("Dpx");
		Dpx_previous = data.at("Sequence").at(obj_name).at("Dpx previous");

		DQx = data.at("Sequence").at(obj_name).at("DQx");
		DQy = data.at("Sequence").at(obj_name).at("DQy");

		longitudinal_transfer = data.at("Sequence").at(obj_name).at("Longitudinal transfer");

		// when �� > ��t (�� > 0), muz (input value) should be > 0
		// when �� < ��t (�� < 0), muz (input value) should be < 0
		muz = data.at("Sequence").at(obj_name).at("Mu z");
		muz_previous = data.at("Sequence").at(obj_name).at("Mu z previous");

	}
	catch (json::exception e)
	{
		//std::cout << e.what() << std::endl;
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}

	phi_x = (mux - mux_previous) * 2 * PassConstant::PI;
	phi_y = (muy - muy_previous) * 2 * PassConstant::PI;
	phi_z = (muz - muz_previous) * 2 * PassConstant::PI;

	if ("drift" == longitudinal_transfer)
	{
		m11_z = 1;
		m12_z = -1 * (1 / (gammat * gammat) - 1 / (gamma * gamma)) * (s - s_previous);
		m21_z = 0;
		m22_z = 1;
	}
	else if ("matrix" == longitudinal_transfer)
	{
		m11_z = cos(phi_z);
		m12_z = sigmaz / dp * sin(phi_z);
		m21_z = -1 * dp / sigmaz * sin(phi_z);
		m22_z = cos(phi_z);
	}
	else
	{
		m11_z = 1;
		m12_z = 0;
		m21_z = 0;
		m22_z = 1;
	}
}

void Twiss::print() {
	auto logger = spdlog::get("logger");

	logger->info("[Twiss] name    = {}, s       = {}", name, s);
	logger->info("[Twiss] Alpha x = {}, Alpha y = {}", alphax, alphay);
	logger->info("[Twiss] Beta  x = {}, Beta  y = {}", betax, betay);
	logger->info("[Twiss] Mu    x = {}, Mu    y = {}", mux, muy);
	logger->info("[Twiss] Alpha x previous = {}, Alpha y previous = {}", alphax_previous, alphay_previous);
	logger->info("[Twiss] Beta  x previous = {}, Beta  y previous = {}", betax_previous, betay_previous);
	logger->info("[Twiss] Mu    x previous = {}, Mu    y previous = {}", mux_previous, muy_previous);

	logger->info("[Twiss] Dx               = {}, Dx previous      = {}", Dx, Dx_previous);
	logger->info("[Twiss] Dpx              = {}, Dpx previous     = {}", Dpx, Dpx_previous);
	logger->info("[Twiss] DQx              = {}, DQy              = {}", DQx, DQy);

	logger->info("[Twiss] Longitudinal transfer = {}", longitudinal_transfer);
	logger->info("[Twiss] Mu   z          = {}", muz);
	logger->info("[Twiss] Mu   z previous = {}", muz);
	logger->info("[Twiss] gamma  = {}, gammat = {}", gamma, gammat);
	logger->info("[Twiss] sigmaz = {}, dp     = {}", sigmaz, dp);
}


void Twiss::execute(int turn) {

	callCuda(hipEventRecord(simTime.start, 0));
	float time_tmp = 0;

	//auto logger = spdlog::get("logger");
	//logger->debug("[Twiss] turn = {}, start running of : {}, s = {}, 6D (logi = {})", turn, name, s, longitudinal_transfer);

	int Np_sur = bunchRef.Np_sur;

	callKernel(
		transfer_matrix_6D << <block_x, thread_x, 0, 0 >> > (dev_particle, Np_sur, circumference,
			betax, betax_previous, alphax, alphax_previous,
			betay, betay_previous, alphay, alphay_previous,
			phi_x, phi_y, DQx * 2 * PassConstant::PI, DQy * 2 * PassConstant::PI,
			Dx, Dx_previous, Dpx, Dpx_previous,
			m11_z, m12_z, m21_z, m22_z)
	);

	//callCuda(hipDeviceSynchronize());
	callCuda(hipEventRecord(simTime.stop, 0));
	callCuda(hipEventSynchronize(simTime.stop));
	callCuda(hipEventElapsedTime(&time_tmp, simTime.start, simTime.stop));
	simTime.twiss += time_tmp;
}


__global__ void transfer_matrix_6D(Particle dev_particle, int Np_sur, double circumference,
	double betax, double betax_previous, double alphax, double alphax_previous,
	double betay, double betay_previous, double alphay, double alphay_previous,
	double phix, double phiy, double DQx, double DQy,
	double Dx, double Dx_previous, double Dpx, double Dpx_previous,
	double m11_z, double m12_z, double m21_z, double m22_z) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	const double c_half = circumference * 0.5;

	const double sqrt_betax_betaxprev = sqrt(betax * betax_previous);
	const double sqrt_betax_de_betaxprev = sqrt(betax / betax_previous);
	const double sqrt_betaxprev_de_betax = sqrt(betax_previous / betax);

	const double sqrt_betay_betayprev = sqrt(betay * betay_previous);
	const double sqrt_betay_de_betayprev = sqrt(betay / betay_previous);
	const double sqrt_betayprev_de_betay = sqrt(betay_previous / betay);

	while (tid < Np_sur)
	{
		int tag = dev_particle.tag[tid];

		if (tag > 0)
		{
			double z1 = dev_particle.z[tid];
			double pz1 = dev_particle.pz[tid];

			double x1 = dev_particle.x[tid] - Dx_previous * pz1;
			double px1 = dev_particle.px[tid] - Dpx_previous * pz1;

			double y1 = dev_particle.y[tid];
			double py1 = dev_particle.py[tid];

			double phi_x = phix + pz1 * DQx;
			double phi_y = phiy + pz1 * DQy;

			double cx = 0, sx = 0, cy = 0, sy = 0;

			sincos(phi_x, &sx, &cx);	// calculate sin(phix) and cos(phix) simultaneously
			sincos(phi_y, &sy, &cy);

			double m11_x = sqrt_betax_de_betaxprev * (cx + alphax_previous * sx);
			double m12_x = sqrt_betax_betaxprev * sx;
			double m21_x = -1 * (1 + alphax * alphax_previous) / sqrt_betax_betaxprev * sx + (alphax_previous - alphax) / sqrt_betax_betaxprev * cx;
			double m22_x = sqrt_betaxprev_de_betax * (cx - alphax * sx);

			double m11_y = sqrt_betay_de_betayprev * (cy + alphay_previous * sy);
			double m12_y = sqrt_betay_betayprev * sy;
			double m21_y = -1 * (1 + alphay * alphay_previous) / sqrt_betay_betayprev * sy + (alphay_previous - alphay) / sqrt_betay_betayprev * cy;
			double m22_y = sqrt_betayprev_de_betay * (cy - alphay * sy);

			double z2 = z1 * m11_z + pz1 * m12_z;
			double pz2 = z1 * m21_z + pz1 * m22_z;

			double x2 = x1 * m11_x + px1 * m12_x + Dx * pz2;
			double px2 = x1 * m21_x + px1 * m22_x + Dpx * pz2;

			double y2 = y1 * m11_y + py1 * m12_y;
			double py2 = y1 * m21_y + py1 * m22_y;

			int over = (z2 > c_half);
			int under = (z2 < -c_half);
			z2 += (under - over) * circumference;

			dev_particle.z[tid] = z2;
			dev_particle.pz[tid] = pz2;
			dev_particle.x[tid] = x2;
			dev_particle.px[tid] = px2;
			dev_particle.y[tid] = y2;
			dev_particle.py[tid] = py2;

			tid += stride;
		}

	}
}
