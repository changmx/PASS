#include "hip/hip_runtime.h"
#include "element.h"

#include <fstream>


MarkerElement::MarkerElement(const Parameter& para, int input_beamId, const Bunch& Bunch, std::string obj_name,
	const ParallelPlan1d& plan1d, TimeEvent& timeevent) :simTime(timeevent), bunchRef(Bunch) {

	commandType = "MarkerElement";
	name = obj_name;
	dev_bunch = Bunch.dev_bunch;

	thread_x = plan1d.get_threads_per_block();
	block_x = plan1d.get_blocks_x();

	Np = Bunch.Np;
	circumference = para.circumference;

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	try
	{
		s = data.at("Sequence").at(obj_name).at("S (m)");
		drift_length = data.at("Sequence").at(obj_name).at("Drift length (m)");

	}
	catch (json::exception e)
	{
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}

}

void MarkerElement::execute(int turn) {

	callCuda(hipEventRecord(simTime.start, 0));
	float time_tmp = 0;

	//auto logger = spdlog::get("logger");
	//logger->info("[Marker Element] run: " + name);

	double gamma = bunchRef.gamma;
	double beta = bunchRef.beta;

	double drift = drift_length;

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, drift);

	callCuda(hipEventRecord(simTime.stop, 0));
	callCuda(hipEventSynchronize(simTime.stop));
	callCuda(hipEventElapsedTime(&time_tmp, simTime.start, simTime.stop));
	simTime.transferElement += time_tmp;
}


SBendElement::SBendElement(const Parameter& para, int input_beamId, const Bunch& Bunch, std::string obj_name,
	const ParallelPlan1d& plan1d, TimeEvent& timeevent) :simTime(timeevent), bunchRef(Bunch) {

	commandType = "SBendElement";
	name = obj_name;
	dev_bunch = Bunch.dev_bunch;

	thread_x = plan1d.get_threads_per_block();
	block_x = plan1d.get_blocks_x();

	Np = Bunch.Np;
	circumference = para.circumference;

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	try
	{
		s = data.at("Sequence").at(obj_name).at("S (m)");
		l = data.at("Sequence").at(obj_name).at("L (m)");
		drift_length = data.at("Sequence").at(obj_name).at("Drift length (m)");
		angle = data.at("Sequence").at(obj_name).at("angle (rad)");
		e1 = data.at("Sequence").at(obj_name).at("e1 (rad)");
		e2 = data.at("Sequence").at(obj_name).at("e2 (rad)");
		hgap = data.at("Sequence").at(obj_name).at("hgap (m)");
		fint = data.at("Sequence").at(obj_name).at("fint");
		fintx = data.at("Sequence").at(obj_name).at("fintx");

		isFieldError = data.at("Sequence").at(obj_name).at("isFieldError");
	}
	catch (json::exception e)
	{
		//std::cout << e.what() << std::endl;
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}
}

void SBendElement::execute(int turn) {

	callCuda(hipEventRecord(simTime.start, 0));
	float time_tmp = 0;

	//auto logger = spdlog::get("logger");
	//logger->info("[SBend Element] run: " + name);

	double gamma = bunchRef.gamma;
	double beta = bunchRef.beta;

	double drift = drift_length;

	double l_used = 0;
	double angle_used = 0;

	if (isFieldError)
	{
		l_used = l / 2;
		angle_used = angle / 2;
	}
	else
	{
		l_used = l;
		angle_used = angle;
	}

	double rho = l_used / angle_used;
	double h = 1 / rho;

	double cx = cos(h * l_used);
	double sx = sin(h * l_used);

	double r11 = cx;
	double r12 = sx / h;
	double r16 = (1 - cx) / (h * beta);
	double r21 = -h * sx;
	double r22 = cx;
	double r26 = sx / beta;
	double r33 = 1;
	double r34 = l_used;
	double r44 = 1;
	double r51 = -sx / beta;
	double r52 = -(1 - cx) / (h * beta);
	double r55 = 1;
	double r56 = l_used / (beta * beta * gamma * gamma) - (h * l_used - sx) / (h * beta * beta);

	double psi1 = e1;
	double psi2 = e2;
	double fint1 = fint;
	double fint2 = fintx;
	double psip1 = psi1 - 2.0 * hgap * h * fint1 / cos(psi1) * (1.0 + pow(sin(psi1), 2));
	double psip2 = psi2 - 2.0 * hgap * h * fint2 / cos(psi2) * (1.0 + pow(sin(psi2), 2));

	double fl21i = h * tan(psi1);
	double fl43i = -h * tan(psip1);
	double fr21i = h * tan(psi2);
	double fr43i = -h * tan(psip2);

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, drift);

	if (isFieldError)
	{
		transfer_dipole_half_left << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta,
			r11, r12, r16, r21, r22, r26, r34, r51, r52, r56, fl21i, fl43i, fr21i, fr43i);

		// transfer multipole error kicker

		transfer_dipole_half_right << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta,
			r11, r12, r16, r21, r22, r26, r34, r51, r52, r56, fl21i, fl43i, fr21i, fr43i);
	}
	else
	{
		transfer_dipole_full << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta,
			r11, r12, r16, r21, r22, r26, r34, r51, r52, r56, fl21i, fl43i, fr21i, fr43i);
	}

	callCuda(hipEventRecord(simTime.stop, 0));
	callCuda(hipEventSynchronize(simTime.stop));
	callCuda(hipEventElapsedTime(&time_tmp, simTime.start, simTime.stop));
	simTime.transferElement += time_tmp;
}


RBendElement::RBendElement(const Parameter& para, int input_beamId, const Bunch& Bunch, std::string obj_name) {
	commandType = "RBendElement";
	name = obj_name;
	dev_bunch = Bunch.dev_bunch;

	Np = Bunch.Np;

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	try
	{
		s = data.at("Sequence").at(obj_name).at("S (m)");
		l = data.at("Sequence").at(obj_name).at("L (m)");
		drift_length = data.at("Sequence").at(obj_name).at("Drift length (m)");
		angle = data.at("Sequence").at(obj_name).at("angle (rad)");
		e1 = data.at("Sequence").at(obj_name).at("e1 (rad)");
		e2 = data.at("Sequence").at(obj_name).at("e2 (rad)");
		hgap = data.at("Sequence").at(obj_name).at("hgap (m)");
		fint = data.at("Sequence").at(obj_name).at("fint");
		fintx = data.at("Sequence").at(obj_name).at("fintx");

	}
	catch (json::exception e)
	{
		//std::cout << e.what() << std::endl;
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}
}

void RBendElement::execute(int turn) {
	auto logger = spdlog::get("logger");
	logger->info("[RBend Element] run: " + name);
}


QuadrupoleElement::QuadrupoleElement(const Parameter& para, int input_beamId, const Bunch& Bunch, std::string obj_name,
	const ParallelPlan1d& plan1d, TimeEvent& timeevent) :simTime(timeevent), bunchRef(Bunch) {

	commandType = "QuadrupoleElement";
	name = obj_name;
	dev_bunch = Bunch.dev_bunch;

	thread_x = plan1d.get_threads_per_block();
	block_x = plan1d.get_blocks_x();

	Np = Bunch.Np;
	circumference = para.circumference;

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	try
	{
		s = data.at("Sequence").at(obj_name).at("S (m)");
		l = data.at("Sequence").at(obj_name).at("L (m)");
		drift_length = data.at("Sequence").at(obj_name).at("Drift length (m)");
		k1 = data.at("Sequence").at(obj_name).at("k1 (m^-2)");
		k1s = data.at("Sequence").at(obj_name).at("k1s (m^-2)");

		isFieldError = data.at("Sequence").at(obj_name).at("isFieldError");
	}
	catch (json::exception e)
	{
		//std::cout << e.what() << std::endl;
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}
}

void QuadrupoleElement::execute(int turn) {
	callCuda(hipEventRecord(simTime.start, 0));
	float time_tmp = 0;

	//auto logger = spdlog::get("logger");
	//logger->info("[Quadrupole Element] run: " + name);

	double gamma = bunchRef.gamma;
	double beta = bunchRef.beta;

	double drift = drift_length;

	double EPSILON = 1e-9;

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, drift);

	if (isFieldError)
	{
		if (abs(k1) > EPSILON && abs(k1s) < EPSILON)	// k1 != 0 && k1s == 0
		{
			transfer_quadrupole_norm << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, k1, l / 2);
			// transfer multipole error kicker
			transfer_quadrupole_norm << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, k1, l / 2);
		}
		else if (abs(k1) < EPSILON && abs(k1s) > EPSILON)	// k1 == 0 && k1s != 0
		{
			transfer_quadrupole_skew << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, k1s, l / 2);
			// transfer multipole error kicker
			transfer_quadrupole_skew << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, k1s, l / 2);
		}
		else
		{
			spdlog::get("logger")->error("[Quadrupole Element] {}: k1 = {}, k1s = {}, there should be and only 1 variable equal to 0",
				name, k1, k1s);
			std::exit(EXIT_FAILURE);
		}
	}
	else
	{
		if (abs(k1) > EPSILON && abs(k1s) < EPSILON)
		{
			transfer_quadrupole_norm << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, k1, l);
		}
		else if (abs(k1) < EPSILON && abs(k1s) > EPSILON)
		{
			transfer_quadrupole_skew << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, k1s, l);
		}
		else
		{
			spdlog::get("logger")->error("[Quadrupole Element] {}: k1 = {}, k1s = {}, there should be and only 1 variable equal to 0",
				name, k1, k1s);
			std::exit(EXIT_FAILURE);
		}
	}

	callCuda(hipEventRecord(simTime.stop, 0));
	callCuda(hipEventSynchronize(simTime.stop));
	callCuda(hipEventElapsedTime(&time_tmp, simTime.start, simTime.stop));
	simTime.transferElement += time_tmp;
}


SextupoleElement::SextupoleElement(const Parameter& para, int input_beamId, const Bunch& Bunch, std::string obj_name,
	const ParallelPlan1d& plan1d, TimeEvent& timeevent) :simTime(timeevent), bunchRef(Bunch) {

	commandType = "SextupoleElement";
	name = obj_name;
	dev_bunch = Bunch.dev_bunch;

	thread_x = plan1d.get_threads_per_block();
	block_x = plan1d.get_blocks_x();

	Np = Bunch.Np;
	circumference = para.circumference;

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	try
	{
		s = data.at("Sequence").at(obj_name).at("S (m)");
		l = data.at("Sequence").at(obj_name).at("L (m)");
		drift_length = data.at("Sequence").at(obj_name).at("Drift length (m)");
		k2 = data.at("Sequence").at(obj_name).at("k2 (m^-3)");
		k2s = data.at("Sequence").at(obj_name).at("k2s (m^-3)");

		isFieldError = data.at("Sequence").at(obj_name).at("isFieldError");
	}
	catch (json::exception e)
	{
		//std::cout << e.what() << std::endl;
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}
}

void SextupoleElement::execute(int turn) {
	callCuda(hipEventRecord(simTime.start, 0));
	float time_tmp = 0;

	//auto logger = spdlog::get("logger");
	//logger->info("[Sextupole Element] run: " + name);

	double gamma = bunchRef.gamma;
	double beta = bunchRef.beta;

	double drift = drift_length;

	double EPSILON = 1e-9;

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, drift + l / 2);

	if (abs(k2) > EPSILON && abs(k2s) < EPSILON)	// k2 != 0 && k2s == 0
	{
		transfer_sextupole_norm << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, k2, l);

	}
	else if (abs(k2) < EPSILON && abs(k2s) > EPSILON)	// k2 == 0 && k2s != 0
	{
		transfer_sextupole_skew << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, k2s, l);
	}
	else
	{
		spdlog::get("logger")->error("[Sextupole Element] {}: ks = {}, kss = {}, there should be and only 1 variable equal to 0",
			name, k2, k2s);
		std::exit(EXIT_FAILURE);
	}

	if (isFieldError)
	{
		// transfer multipole error kicker
	}

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, l / 2);
}


OctupoleElement::OctupoleElement(const Parameter& para, int input_beamId, const Bunch& Bunch, std::string obj_name,
	const ParallelPlan1d& plan1d, TimeEvent& timeevent) :simTime(timeevent), bunchRef(Bunch) {

	commandType = "OctupoleElement";
	name = obj_name;
	dev_bunch = Bunch.dev_bunch;

	thread_x = plan1d.get_threads_per_block();
	block_x = plan1d.get_blocks_x();

	Np = Bunch.Np;
	circumference = para.circumference;

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	try
	{
		s = data.at("Sequence").at(obj_name).at("S (m)");
		l = data.at("Sequence").at(obj_name).at("L (m)");
		drift_length = data.at("Sequence").at(obj_name).at("Drift length (m)");
		k3 = data.at("Sequence").at(obj_name).at("k3 (m^-4)");
		k3s = data.at("Sequence").at(obj_name).at("k3s (m^-4)");

		isFieldError = data.at("Sequence").at(obj_name).at("isFieldError");
	}
	catch (json::exception e)
	{
		//std::cout << e.what() << std::endl;
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}
}


void OctupoleElement::execute(int turn) {
	callCuda(hipEventRecord(simTime.start, 0));
	float time_tmp = 0;

	//auto logger = spdlog::get("logger");
	//logger->info("[Sextupole Element] run: " + name);

	double gamma = bunchRef.gamma;
	double beta = bunchRef.beta;

	double drift = drift_length;

	double EPSILON = 1e-9;

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, drift + l / 2);

	if (abs(k3) > EPSILON && abs(k3s) < EPSILON)	// k3 != 0 && k3s == 0
	{
		transfer_octupole_norm << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, k3, l);

	}
	else if (abs(k3) < EPSILON && abs(k3s) > EPSILON)	// k3 == 0 && k3s != 0
	{
		transfer_octupole_skew << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, k3s, l);
	}
	else
	{
		spdlog::get("logger")->error("[Octupole Element] {}: ks = {}, kss = {}, there should be and only 1 variable equal to 0",
			name, k3, k3s);
		std::exit(EXIT_FAILURE);
	}

	if (isFieldError)
	{
		// transfer multipole error kicker
	}

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, l / 2);
}


HKickerElement::HKickerElement(const Parameter& para, int input_beamId, const Bunch& Bunch, std::string obj_name,
	const ParallelPlan1d& plan1d, TimeEvent& timeevent) :simTime(timeevent), bunchRef(Bunch) {

	commandType = "HKickerElement";
	name = obj_name;
	dev_bunch = Bunch.dev_bunch;

	thread_x = plan1d.get_threads_per_block();
	block_x = plan1d.get_blocks_x();

	Np = Bunch.Np;
	circumference = para.circumference;

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	try
	{
		s = data.at("Sequence").at(obj_name).at("S (m)");
		l = data.at("Sequence").at(obj_name).at("L (m)");
		drift_length = data.at("Sequence").at(obj_name).at("Drift length (m)");
		kick = data.at("Sequence").at(obj_name).at("kick");

		isFieldError = data.at("Sequence").at(obj_name).at("isFieldError");
	}
	catch (json::exception e)
	{
		//std::cout << e.what() << std::endl;
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}
}


void HKickerElement::execute(int turn) {
	callCuda(hipEventRecord(simTime.start, 0));
	float time_tmp = 0;

	//auto logger = spdlog::get("logger");
	//logger->info("[Sextupole Element] run: " + name);

	double gamma = bunchRef.gamma;
	double beta = bunchRef.beta;

	double drift = drift_length;

	double EPSILON = 1e-9;

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, drift + l / 2);

	transfer_hkicker << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, kick);

	if (isFieldError)
	{
		// transfer multipole error kicker
	}

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, l / 2);
}


VKickerElement::VKickerElement(const Parameter& para, int input_beamId, const Bunch& Bunch, std::string obj_name,
	const ParallelPlan1d& plan1d, TimeEvent& timeevent) :simTime(timeevent), bunchRef(Bunch) {

	commandType = "VKickerElement";
	name = obj_name;
	dev_bunch = Bunch.dev_bunch;

	thread_x = plan1d.get_threads_per_block();
	block_x = plan1d.get_blocks_x();

	Np = Bunch.Np;
	circumference = para.circumference;

	using json = nlohmann::json;
	std::ifstream jsonFile(para.path_input_para[input_beamId]);
	json data = json::parse(jsonFile);

	try
	{
		s = data.at("Sequence").at(obj_name).at("S (m)");
		l = data.at("Sequence").at(obj_name).at("L (m)");
		drift_length = data.at("Sequence").at(obj_name).at("Drift length (m)");
		kick = data.at("Sequence").at(obj_name).at("kick");

		isFieldError = data.at("Sequence").at(obj_name).at("isFieldError");
	}
	catch (json::exception e)
	{
		//std::cout << e.what() << std::endl;
		spdlog::get("logger")->error(e.what());
		std::exit(EXIT_FAILURE);
	}
}


void VKickerElement::execute(int turn) {
	callCuda(hipEventRecord(simTime.start, 0));
	float time_tmp = 0;

	//auto logger = spdlog::get("logger");
	//logger->info("[Sextupole Element] run: " + name);

	double gamma = bunchRef.gamma;
	double beta = bunchRef.beta;

	double drift = drift_length;

	double EPSILON = 1e-9;

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, drift + l / 2);

	transfer_vkicker << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, kick);

	if (isFieldError)
	{
		// transfer multipole error kicker
	}

	transfer_drift << <block_x, thread_x, 0, 0 >> > (dev_bunch, Np, beta, gamma, l / 2);
}


__global__ void transfer_drift(Particle* dev_bunch, int Np,
	double beta, double gamma, double drift_length) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double r12 = drift_length;
	double r34 = drift_length;
	double r56 = drift_length / (beta * beta * gamma * gamma);

	double tau0 = 0, tau1 = 0;	// tau = z/beta - ct(=0) = z/beta
	double pt0 = 0;	// pt = DeltaE/(P0*c) = beta*DeltaP/P0

	while (tid < Np) {

		tau0 = dev_bunch[tid].z / beta;
		pt0 = dev_bunch[tid].pz * beta;

		dev_bunch[tid].x += r12 * dev_bunch[tid].px;
		dev_bunch[tid].y += r34 * dev_bunch[tid].py;

		tau1 = tau0 + r56 * pt0;
		dev_bunch[tid].z = tau1 * beta;

		tid += stride;
	}
}

__global__ void transfer_dipole_full(Particle* dev_bunch, int Np, double beta,
	double r11, double r12, double r16, double r21, double r22, double r26,
	double r34, double r51, double r52, double r56,
	double fl21i, double fl43i, double fr21i, double fr43i) {

	// I would like to express my gratitude to Dr.Ren Hang(renhang@impcas.ac.cn)
	// for providing the code for dipole transfer 

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double x1 = 0, px1 = 0, y1 = 0, py1 = 0;
	double x2 = 0, px2 = 0, y2 = 0, py2 = 0;
	double x3 = 0, px3 = 0, y3 = 0, py3 = 0;
	double tau0 = 0, tau1 = 0, tau2 = 0, tau3 = 0;	// tau = z/beta - ct(=0) = z/beta
	double pt0 = 0, pt1 = 0, pt2 = 0, pt3 = 0;	// pt = DeltaE/(P0*c) = beta*DeltaP/P0

	double fl21 = 0, fl43 = 0, fr21 = 0, fr43 = 0;

	double d = 0;	// about power error

	while (tid < Np) {

		tau0 = dev_bunch[tid].z / beta;
		pt0 = dev_bunch[tid].pz * beta;

		double fl21 = fl21i / (1 + pt0 / beta);
		double fl43 = fl43i / (1 + pt0 / beta);
		double fr21 = fr21i / (1 + pt0 / beta);
		double fr43 = fr43i / (1 + pt0 / beta);

		// apply the influence of left fringe field
		x1 = dev_bunch[tid].x;
		px1 = dev_bunch[tid].px + fl21 * x1;
		y1 = dev_bunch[tid].y;
		py1 = dev_bunch[tid].py + fl43 * y1;
		tau1 = tau0;
		pt1 = pt0 + d;

		// apply the influence of dipole
		x2 = r11 * x1 + r12 * px1 + r16 * pt1;
		px2 = r21 * x1 + r22 * px1 + r26 * pt1;
		y2 = y1 + r34 * py1;
		py2 = py1;
		tau2 = r51 * x1 + r52 * px1 + tau1 + r56 * pt1;
		pt2 = pt1;

		// apply the influece of right fringe field
		x3 = x2;
		px3 = px2 + fr21 * x2;
		y3 = y2;
		py3 = py2 + fr43 * y2;
		tau3 = tau2;
		pt3 = pt2 - d;

		dev_bunch[tid].x = x3;
		dev_bunch[tid].px = px3;
		dev_bunch[tid].y = y3;
		dev_bunch[tid].py = py3;
		dev_bunch[tid].z = tau3 * beta;
		dev_bunch[tid].pz = pt3 / beta;

		tid += stride;
	}

}


__global__ void transfer_dipole_half_left(Particle* dev_bunch, int Np, double beta,
	double r11, double r12, double r16, double r21, double r22, double r26,
	double r34, double r51, double r52, double r56,
	double fl21i, double fl43i, double fr21i, double fr43i) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double x1 = 0, px1 = 0, y1 = 0, py1 = 0;
	double x2 = 0, px2 = 0, y2 = 0, py2 = 0;
	double x3 = 0, px3 = 0, y3 = 0, py3 = 0;
	double tau0 = 0, tau1 = 0, tau2 = 0, tau3 = 0;	// tau = z/beta - ct(=0) = z/beta
	double pt0 = 0, pt1 = 0, pt2 = 0, pt3 = 0;	// pt = DeltaE/(P0*c) = beta*DeltaP/P0

	double fl21 = 0, fl43 = 0, fr21 = 0, fr43 = 0;

	double d = 0;	// about power error

	while (tid < Np) {

		tau0 = dev_bunch[tid].z / beta;
		pt0 = dev_bunch[tid].pz * beta;

		double fl21 = fl21i / (1 + pt0 / beta);
		double fl43 = fl43i / (1 + pt0 / beta);
		double fr21 = fr21i / (1 + pt0 / beta);
		double fr43 = fr43i / (1 + pt0 / beta);

		// apply the influence of left fringe field
		x1 = dev_bunch[tid].x;
		px1 = dev_bunch[tid].px + fl21 * x1;
		y1 = dev_bunch[tid].y;
		py1 = dev_bunch[tid].py + fl43 * y1;
		tau1 = tau0;
		pt1 = pt0 + d;

		// apply the influence of dipole
		x2 = r11 * x1 + r12 * px1 + r16 * pt1;
		px2 = r21 * x1 + r22 * px1 + r26 * pt1;
		y2 = y1 + r34 * py1;
		py2 = py1;
		tau2 = r51 * x1 + r52 * px1 + tau1 + r56 * pt1;
		pt2 = pt1;

		// no influece of right fringe field
		x3 = x2;
		px3 = px2;
		y3 = y2;
		py3 = py2;
		tau3 = tau2;
		pt3 = pt2 - d;

		dev_bunch[tid].x = x3;
		dev_bunch[tid].px = px3;
		dev_bunch[tid].y = y3;
		dev_bunch[tid].py = py3;
		dev_bunch[tid].z = tau3 * beta;
		dev_bunch[tid].pz = pt3 / beta;

		tid += stride;
	}

}


__global__ void transfer_dipole_half_right(Particle* dev_bunch, int Np, double beta,
	double r11, double r12, double r16, double r21, double r22, double r26,
	double r34, double r51, double r52, double r56,
	double fl21i, double fl43i, double fr21i, double fr43i) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double x1 = 0, px1 = 0, y1 = 0, py1 = 0;
	double x2 = 0, px2 = 0, y2 = 0, py2 = 0;
	double x3 = 0, px3 = 0, y3 = 0, py3 = 0;
	double tau0 = 0, tau1 = 0, tau2 = 0, tau3 = 0;	// tau = z/beta - ct(=0) = z/beta
	double pt0 = 0, pt1 = 0, pt2 = 0, pt3 = 0;	// pt = DeltaE/(P0*c) = beta*DeltaP/P0

	double fl21 = 0, fl43 = 0, fr21 = 0, fr43 = 0;

	double d = 0;	// about power error

	while (tid < Np) {

		tau0 = dev_bunch[tid].z / beta;
		pt0 = dev_bunch[tid].pz * beta;

		double fl21 = fl21i / (1 + pt0 / beta);
		double fl43 = fl43i / (1 + pt0 / beta);
		double fr21 = fr21i / (1 + pt0 / beta);
		double fr43 = fr43i / (1 + pt0 / beta);

		// no influence of left fringe field
		x1 = dev_bunch[tid].x;
		px1 = dev_bunch[tid].px;
		y1 = dev_bunch[tid].y;
		py1 = dev_bunch[tid].py;
		tau1 = tau0;
		pt1 = pt0 + d;

		// apply the influence of dipole
		x2 = r11 * x1 + r12 * px1 + r16 * pt1;
		px2 = r21 * x1 + r22 * px1 + r26 * pt1;
		y2 = y1 + r34 * py1;
		py2 = py1;
		tau2 = r51 * x1 + r52 * px1 + tau1 + r56 * pt1;
		pt2 = pt1;

		// apply the influece of right fringe field
		x3 = x2;
		px3 = px2 + fr21 * x2;
		y3 = y2;
		py3 = py2 + fr43 * y2;
		tau3 = tau2;
		pt3 = pt2 - d;

		dev_bunch[tid].x = x3;
		dev_bunch[tid].px = px3;
		dev_bunch[tid].y = y3;
		dev_bunch[tid].py = py3;
		dev_bunch[tid].z = tau3 * beta;
		dev_bunch[tid].pz = pt3 / beta;

		tid += stride;
	}

}


__global__ void transfer_quadrupole_norm(Particle* dev_bunch, int Np, double beta,
	double k1, double l) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double x0 = 0, px0 = 0, y0 = 0, py0 = 0;
	double x1 = 0, px1 = 0, y1 = 0, py1 = 0;

	double tau0 = 0, tau1 = 0;	// tau = z/beta - ct(=0) = z/beta
	double pt0 = 0;	// pt = DeltaE/(P0*c) = beta*DeltaP/P0

	double k1_chrom = 0, omega = 0;
	double cx = 0, sx = 0, chx = 0, shx = 0;
	double r11 = 0, r12 = 0, r21 = 0, r22 = 0, r33 = 0, r34 = 0, r43 = 0, r44 = 0, r56 = 0;

	while (tid < Np) {

		x0 = dev_bunch[tid].x;
		px0 = dev_bunch[tid].px;
		y0 = dev_bunch[tid].y;
		py0 = dev_bunch[tid].py;
		tau0 = dev_bunch[tid].z / beta;
		pt0 = dev_bunch[tid].pz * beta;

		k1_chrom = k1 / (1 + pt0 / beta);
		omega = sqrt(abs(k1_chrom));

		cx = cos(omega * l);
		sx = sin(omega * l);
		chx = cosh(omega * l);
		shx = sinh(omega * l);

		if (k1_chrom > 0) {
			r11 = cx;
			r12 = sx / omega;
			r21 = -omega * sx;
			r22 = cx;
			r33 = chx;
			r34 = shx / omega;
			r43 = omega * shx;
			r44 = chx;
			r56 = l * (1 / (beta * beta) - 1);
		}
		else {
			r11 = chx;
			r12 = shx / omega;
			r21 = omega * shx;
			r22 = chx;
			r33 = cx;
			r34 = sx / omega;
			r43 = -omega * sx;
			r44 = cx;
			r56 = l * (1 / (beta * beta) - 1);
		}

		x1 = r11 * x0 + r12 * px0;
		px1 = r21 * x0 + r22 * px0;
		y1 = r33 * y0 + r34 * py0;
		py1 = r43 * y0 + r44 * py0;
		tau1 = tau0 + r56 * pt0;

		dev_bunch[tid].x = x1;
		dev_bunch[tid].px = px1;
		dev_bunch[tid].y = y1;
		dev_bunch[tid].py = py1;
		dev_bunch[tid].z = tau1 * beta;

		tid += stride;
	}
}


__global__ void transfer_quadrupole_skew(Particle* dev_bunch, int Np, double beta,
	double k1s, double l) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double x0 = 0, px0 = 0, y0 = 0, py0 = 0;
	double x1 = 0, px1 = 0, y1 = 0, py1 = 0;

	double tau0 = 0, tau1 = 0;	// tau = z/beta - ct(=0) = z/beta
	double pt0 = 0;	// pt = DeltaE/(P0*c) = beta*DeltaP/P0

	double k1s_chrom = 0, omega = 0;
	double cx = 0, sx = 0, chx = 0, shx = 0;
	double cp = 0, cm = 0, sp = 0, sm = 0;
	double r11 = 0, r12 = 0, r13 = 0, r14 = 0, r21 = 0, r22 = 0, r23 = 0, r24 = 0;
	double r31 = 0, r32 = 0, r33 = 0, r34 = 0, r41 = 0, r42 = 0, r43 = 0, r44 = 0;
	double r56 = 0;

	while (tid < Np) {

		x0 = dev_bunch[tid].x;
		px0 = dev_bunch[tid].px;
		y0 = dev_bunch[tid].y;
		py0 = dev_bunch[tid].py;
		tau0 = dev_bunch[tid].z / beta;
		pt0 = dev_bunch[tid].pz * beta;

		k1s_chrom = k1s / (1 + pt0 / beta);
		omega = sqrt(abs(k1s_chrom));

		cx = cos(omega * l);
		sx = sin(omega * l);
		chx = cosh(omega * l);
		shx = sinh(omega * l);

		cp = (cx + chx) / 2;
		cm = (cx - chx) / 2;
		sp = (sx + shx) / 2;
		sm = (sx - shx) / 2;


		if (k1s_chrom > 0) {
			r11 = cp;
			r12 = sp / omega;
			r13 = cm;
			r14 = sm / omega;
			r21 = -omega * sm;
			r22 = cp;
			r23 = -omega * sp;
			r24 = cm;
			r31 = cm;
			r32 = sm / omega;
			r33 = cp;
			r34 = sp / omega;
			r41 = -omega * sp;
			r42 = cm;
			r43 = -omega * sm;
			r44 = cp;
			r56 = l * (1 / (beta * beta) - 1);
		}
		else {
			r11 = cp;
			r12 = sp / omega;
			r13 = -cm;
			r14 = -sm / omega;
			r21 = -omega * sm;
			r22 = cp;
			r23 = omega * sp;
			r24 = -cm;
			r31 = -cm;
			r32 = -sm / omega;
			r33 = cp;
			r34 = sp / omega;
			r41 = omega * sp;
			r42 = -cm;
			r43 = -omega * sm;
			r44 = cp;
			r56 = l * (1 / (beta * beta) - 1);
		}

		x1 = r11 * x0 + r12 * px0 + r13 * y0 + r14 * py0;
		px1 = r21 * x0 + r22 * px0 + r23 * y0 + r24 * py0;
		y1 = r31 * x0 + r32 * px0 + r33 * y0 + r34 * py0;
		py1 = r41 * x0 + r42 * px0 + r43 * y0 + r44 * py0;
		tau1 = tau0 + r56 * pt0;

		dev_bunch[tid].x = x1;
		dev_bunch[tid].px = px1;
		dev_bunch[tid].y = y1;
		dev_bunch[tid].py = py1;
		dev_bunch[tid].z = tau1 * beta;

		tid += stride;
	}
}


__global__ void transfer_sextupole_norm(Particle* dev_bunch, int Np, double beta,
	double k2, double l) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double x0 = 0, px0 = 0, y0 = 0, py0 = 0;
	double k2_chrom = 0;

	while (tid < Np) {

		x0 = dev_bunch[tid].x;
		px0 = dev_bunch[tid].px;
		y0 = dev_bunch[tid].y;
		py0 = dev_bunch[tid].py;

		k2_chrom = k2 / (1 + dev_bunch[tid].pz);

		dev_bunch[tid].px += -0.5 * k2_chrom * l * (x0 * x0 - y0 * y0);
		dev_bunch[tid].py += k2_chrom * l * x0 * y0;

		tid += stride;
	}
}


__global__ void transfer_sextupole_skew(Particle* dev_bunch, int Np, double beta,
	double k2s, double l) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double x0 = 0, px0 = 0, y0 = 0, py0 = 0;
	double k2s_chrom = 0;

	while (tid < Np) {

		x0 = dev_bunch[tid].x;
		px0 = dev_bunch[tid].px;
		y0 = dev_bunch[tid].y;
		py0 = dev_bunch[tid].py;

		k2s_chrom = k2s / (1 + dev_bunch[tid].pz);

		dev_bunch[tid].px += k2s_chrom * l * x0 * y0;
		dev_bunch[tid].py += 0.5 * k2s_chrom * l * (x0 * x0 - y0 * y0);

		tid += stride;
	}
}


__global__ void transfer_octupole_norm(Particle* dev_bunch, int Np, double beta,
	double k3, double l) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double x0 = 0, px0 = 0, y0 = 0, py0 = 0;
	double k3_chrom = 0;

	while (tid < Np) {

		x0 = dev_bunch[tid].x;
		px0 = dev_bunch[tid].px;
		y0 = dev_bunch[tid].y;
		py0 = dev_bunch[tid].py;

		k3_chrom = k3 / (1 + dev_bunch[tid].pz);

		dev_bunch[tid].px += -1.0 / 6 * k3_chrom * l * (pow(x0, 3) - 3 * x0 * pow(y0, 2));
		dev_bunch[tid].py += 1.0 / 6 * k3_chrom * l * (3 * pow(x0, 2) * y0 - pow(y0, 3));

		tid += stride;
	}
}


__global__ void transfer_octupole_skew(Particle* dev_bunch, int Np, double beta,
	double k3s, double l) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double x0 = 0, px0 = 0, y0 = 0, py0 = 0;
	double k3s_chrom = 0;

	while (tid < Np) {

		x0 = dev_bunch[tid].x;
		px0 = dev_bunch[tid].px;
		y0 = dev_bunch[tid].y;
		py0 = dev_bunch[tid].py;

		k3s_chrom = k3s / (1 + dev_bunch[tid].pz);

		dev_bunch[tid].px += 1.0 / 6 * k3s_chrom * l * (3 * pow(x0, 2) * y0 - pow(y0, 3));
		dev_bunch[tid].py += 1.0 / 6 * k3s_chrom * l * (pow(x0, 3) - 3 * x0 * pow(y0, 2));

		tid += stride;
	}
}


__global__ void transfer_hkicker(Particle* dev_bunch, int Np, double beta,
	double kick) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double x0 = 0, px0 = 0;
	double kick_chrom = 0;

	while (tid < Np) {

		x0 = dev_bunch[tid].x;
		px0 = dev_bunch[tid].px;

		kick_chrom = kick / (1 + dev_bunch[tid].pz);

		dev_bunch[tid].px += kick_chrom;

		tid += stride;
	}
}


__global__ void transfer_vkicker(Particle* dev_bunch, int Np, double beta,
	double kick) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	double y0 = 0, py0 = 0;
	double kick_chrom = 0;

	while (tid < Np) {

		y0 = dev_bunch[tid].y;
		py0 = dev_bunch[tid].py;

		kick_chrom = kick / (1 + dev_bunch[tid].pz);

		dev_bunch[tid].py += kick_chrom;

		tid += stride;
	}
}